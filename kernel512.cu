#include "hip/hip_runtime.h"
﻿///////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////
//В строках "в районе" 30-й - начальные параметры. В строках около 150-й нужно указать путь к файлам с функциями(либо положить в ту же директорию, где основной, и убрать путь к ним)
//если компилятор не знает M_PI, то вернуть строку (с дефайном MPI) из комментария
//Если у компа меньше 64 гигов оперативы(48, на самом деле, тоже должно подойти), то лучше не запускать с сетками больше 256
//Сетки с размерами, не являющимися степенями двойки, могут работать, но НЕ ТЕСТИРОВАЛИСЬ
//Директория для выходных файлов задается в константах
///////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////


#include "hip/hip_runtime.h"
#include ""
#include "cuda_error_hadling.h"
#include <cstdio>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <ctime>
#include <fstream>
#include <cstring>

//#define M_PI 3.1415926535897932384626433832795
#define Gl 0.01
#define nu 0.025
#define eta 0.025
#define N 256
#define h 2.0*M_PI / double(N)
#define Cfl  0.1
#define Cfl2 0.1 //цфл поглощения, < 1/8
#define Omx 0.0 // омега икс
#define Omy 0.0 // омега игрек
#define Omz 0.0 // омега зет
#define B0x 0.1
#define B0y 0.0
#define B0z 0.0
#define lambda (double(N)*double(N)*double(N))
#define kmin 1  //левая граница начального спектра
#define kmax 10  //правая граница начального спектра
#define EForse 0.0001 //энергия, вливаемая в dtmax во всем промежутке по k(в случае с нелинейным спектром - энергия до свертки)
#define dtmax 0.01
using namespace std;

hipError_t cuerr;
const char OutputDirectory[80] = "/mnt/data2/256/";//директория
const double Time = 40;//расчетное время
const bool vivodphis = 0;//выводить или нет
const bool vivodspec = 1;//выводить или нет
const bool vivodhelicity = 1;//выводить или нет
const bool useforcing = 0;//форсинг(КОД ПОД ФОРСИНГ ЕСТЬ, НО НЕ ТЕСТИРОВАЛСЯ, могут быть косяки)
const int numberofpreviousforcing = 5000;
double dt = 1;//тут число просто чтобы если что-то не так, увидеть, что он не меняет время
double Nk = 0.1;//шаг времени вывода спектров
double dNk = Nk;
double Nk2 = 1.0;//шаг времени вывода скорости
double dNk2 = Nk2;
int NNk = 10;//это для нумерации выходных файлов
int NNk2 = 10;
int NNN = 0;//количество шагов по времени


hipfftDoubleReal *UX, *UY, *UZ;//в физdevice
hipfftDoubleReal *h_UX, *h_UY, *h_UZ;//в физhost

hipfftDoubleComplex *UXv, *UYv, *UZv;//в фурье пространстве device
hipfftDoubleComplex *h_UXv, *h_UYv, *h_UZv;//в фурье пространстве host

hipfftDoubleComplex *dUv;//в фурье(временные переменные и для поля и для скоростей)

hipfftDoubleReal *d1U, *d2U, *d3U, *d4U, *d5U, *d6U;//в физ(временные переменные и для поля и для скоростей) device
hipfftDoubleReal *h_d2U,*h_d3U,*h_d4U,*h_d5U, *h_d6U;//в физ(временные переменные и для поля и для скоростей) host

hipfftDoubleReal *AiD;//в физ(временные переменные)

hipfftDoubleComplex *AXv, *AYv, *AZv;//в фурье(временные переменные) device
hipfftDoubleComplex *h_AXv, *h_AYv, *h_AZv;//в фурье(временные переменные) host

hipfftDoubleComplex *P;//давление в фурье device
hipfftDoubleComplex *h_P;//давление в фурье host

hipfftDoubleComplex *F;//правая часть уравнения device
hipfftDoubleComplex *h_F;//правая часть уравнения host

hipfftDoubleReal *TCH; //dt,cfl,h в видеопамяти
hipfftDoubleReal *tch; //dt,cfl,h в оперативе

hipfftDoubleComplex *U1, *U2, *U3; //временные переменные(промежуточные значения) device
hipfftDoubleComplex *h_U1, *h_U2, *h_U3; //временные переменные(промежуточные значения) host

hipfftDoubleComplex *U1n, *U2n, *U3n; //временные переменные(конечные значения на шаге) device
hipfftDoubleComplex *h_U1n, *h_U2n, *h_U3n; //временные переменные(конечные значения на шаге) host

hipfftDoubleComplex *bx, *by, *bz;//(в оперативе)

hipfftDoubleReal *BX, *BY, *BZ;//в физ device
hipfftDoubleReal *h_BX, *h_BY, *h_BZ;//в физ host

hipfftDoubleComplex *BXv, *BYv, *BZv;//в фурье пространстве device
hipfftDoubleComplex *h_BXv, *h_BYv, *h_BZv;//в фурье пространстве host

hipfftDoubleComplex *DXv, *DYv, *DZv;//в фурье(временные переменные) device
hipfftDoubleComplex *h_DXv, *h_DYv, *h_DZv;//в фурье(временные переменные) host

hipfftDoubleComplex *B1, *B2, *B3; //временные переменные(промежуточные значения)device
hipfftDoubleComplex *h_B1, *h_B2, *h_B3; //временные переменные(промежуточные значения)host

hipfftDoubleComplex *B1n, *B2n, *B3n; //временные переменные(конечные значения на шаге)device
hipfftDoubleComplex *h_B1n, *h_B2n, *h_B3n; //временные переменные(конечные значения на шаге)host
const int csize = N*N*N * sizeof(hipfftDoubleReal);
const int csizeIm = N*N*(N/2+1) * sizeof(hipfftDoubleComplex);
FILE*	out1;
FILE*	out2;
FILE*	out3;

int *WN;
int *Kf;//оператива
hipfftDoubleReal *abc;//оператива
int num1, num2;
hipfftDoubleReal phix, phiy, phiz, Estep;
int kx, ky, kz;
double kmod;
hipfftDoubleReal ABC1, ABC2, ABC3;

hipfftDoubleReal *ex, *ey, *ez;
hipfftDoubleReal *ebx, *eby, *ebz;
hipfftDoubleReal *Ukx, *Uky, *Ukz;//под энергию(3дспектр) device
hipfftDoubleReal *h_Ukx, *h_Uky, *h_Ukz;//host
hipfftDoubleReal *ukx, *uky, *ukz;//под энергию(3дспектр) в оперативе
hipfftDoubleReal *bkx, *bky, *bkz;//под энергию(3дспектр) в оперативе

int kxyz;


hipfftHandle plan;
hipfftHandle planinverse;

const int K1 = 8;
const int K2 = K1;
const int K3 = K1;

dim3 dimBlock(K1, K2, K3);//Размер блока
dim3 dimGrid(N / K1, N / K2, N / K3); //количество блоков для действительных ядер
dim3 dimGridZ(N / K1, N / K2, N / (2 * K3));//количество юлоков для комплексных ядер



///////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////
//ТУТ НАДО УКАЗАТЬ ВЕРНЫЕ ПУТИ К ФАЙЛАМ, в них лежат функции, без них нифига работать не будет
//как вариант, можно кинуть файлы в ту же папку, где лежит основной и убрать пути к ним, вроде как, работает(но зависит от компилятора, насколько я помню)
///////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////

#include "/home/yst/data/FunctionsNelin.h"
#include "/home/yst/data/FunctionsForce.h"
#include "/home/yst/data/Helicity_Functions.h"


int main()
{

				///////////////////////////////////////////////
				// информация о всех CUDA GPU в системе
				///////////////////////////////////////////////
//	setlocale(LC_ALL, "Russian");
	//hipDeviceProp_t count;
	int c;
	hipGetDeviceCount(&c);
	/*cout << "Количество GPU: " << c << endl;
	for (int i = 0; i < c; i++)
	{
		hipGetDeviceProperties(&count, 0);
		cout << "GPU №" << i + 1 << ": " << count.name << endl;
		cout << "Тактовая частота: " << count.clockRate / 1000 << " ГГц" << endl;
		cout << "Глобальная память(всего): " << count.totalGlobalMem / 1024.0 / 1024.0 << " МБ" << endl;
		cout << "Константная память(всего): " << count.totalConstMem / 1024.0 << " КБ" << endl;
		cout << "Количество мультипроцессоров: " << count.multiProcessorCount << endl;
		cout << "Разделяемая память на один МП: " << count.sharedMemPerBlock / 1024.0 << " КБ" << endl;
		cout << "Нитей в варпе: " << count.warpSize << endl;
		cout << "Макс количество нитей в блоке: " << count.maxThreadsPerBlock << endl;
		cout << "Макс количество нитей по измерениям: " << count.maxThreadsDim[0] << ' ' << count.maxThreadsDim[1] << ' ' << count.maxThreadsDim[2] << endl;
		cout << "Максимальные размеры сетки: " << count.maxGridSize[0] << ' ' << count.maxGridSize[1] << ' ' << count.maxGridSize[2] << endl;
		cout << "Может ли отображать память CPU на пространство CUDA-устройства: " << count.canMapHostMemory << endl;
		cout << "Является ли интегрированным: " << count.integrated << endl;

	}*/
	//cout << "Введите номер выбранного для расчетов GPU (n<=" << c <<")"<< endl;
	//cin >> c;
	c = 0; //пока что по умолчанию гпу=1
		   //выбираем GPU
	hipSetDevice(c);


				///////////////////////////////////////////////
				//А тут уже основной кусок проги
				///////////////////////////////////////////////
	double T = 0;
	unsigned int start_time = clock()/CLOCKS_PER_SEC;// начальное время ввода данных
	srand(time(0));//включаем рандомизацию

				///////////////////////////////////////////////
				//создаём план
				///////////////////////////////////////////////

	hipfftPlan3d(&plan, N, N, N, HIPFFT_D2Z);
	hipfftPlan3d(&planinverse, N, N, N, HIPFFT_Z2D);



	ios_base::sync_with_stdio(0);
	char file[160];
	char dir[10];

	unsigned int end_time = clock()/CLOCKS_PER_SEC;//конечное время ввода данных
	//cout << "download data in " << (end_time - start_time)<< " sek" << endl;


				///////////////////////////////////////////////
				//расчёт всех возможных вариантов волновых чисел(естественно, квадратов волновых чисел)
				///////////////////////////////////////////////
	int Nl = 0; bool dn = 1; int si2;

	WN = (int *)malloc(N*N*N * sizeof(int));

	for (int ix = 0; ix < N / 3; ix += 1)
		for (int iy = 0; iy < N / 3; iy += 1)
			for (int iz = 0; iz < N / 3; iz += 1)
			{
				si2 = ix*ix + iy*iy + iz*iz;
				dn = 1;
				for (int i = 0; i < Nl; i++) if ((WN[i] == si2)||(si2>= (N / 3)*(N / 3))) dn = 0;
				if (dn)
				{
					WN[Nl] = si2;
					Nl++;
				}
			}

	qsort(WN, Nl, sizeof(int), compare);
	
	strcpy(file, OutputDirectory);
	strncat(file, "K.dat", 20);
	out1 = fopen(file, "w+");
	for (int i = 0; i < Nl; i++)
	{
		fprintf(out1, "%i\n", WN[i]);
	}
	fclose(out1);

	hipHostAlloc((void **)&h_Ukx,
				   N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal),
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&Ukx,h_Ukx,0);

	hipHostAlloc((void **)&h_Uky,
				   N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal),
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&Uky,h_Uky,0);

	hipHostAlloc((void **)&h_Ukz,
				   N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal),
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&Ukz,h_Ukz,0);


	ukx = (hipfftDoubleReal *)malloc(N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal));
	uky = (hipfftDoubleReal *)malloc(N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal));
	ukz = (hipfftDoubleReal *)malloc(N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal));
	bkx = (hipfftDoubleReal *)malloc(N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal));
	bky = (hipfftDoubleReal *)malloc(N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal));
	bkz = (hipfftDoubleReal *)malloc(N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal));
	ex = (hipfftDoubleReal *)malloc(Nl * sizeof(hipfftDoubleReal));
	ey = (hipfftDoubleReal *)malloc(Nl * sizeof(hipfftDoubleReal));
	ez = (hipfftDoubleReal *)malloc(Nl * sizeof(hipfftDoubleReal));
	ebx = (hipfftDoubleReal *)malloc(Nl * sizeof(hipfftDoubleReal));
	eby = (hipfftDoubleReal *)malloc(Nl * sizeof(hipfftDoubleReal));
	ebz = (hipfftDoubleReal *)malloc(Nl * sizeof(hipfftDoubleReal));
	int id1, id2, id3, id4;

				///////////////////////////////////////////////
				//расчёт всех возможных вариантов волновых чисел k_x, k_y, k_z для задания начальных условий(в пределах k_x, k_y, k_z)
				///////////////////////////////////////////////

	int Nf = 0;
	double imax = kmax*sqrt(3);
	for (int ix = 1; ix < imax; ix += 1)
		for (int iy = 1; iy < imax; iy += 1)
			for (int iz = 1; iz < imax; iz += 1)
				if ((ix*ix + iy*iy + iz*iz >= kmin*kmin) && (ix*ix + iy*iy + iz*iz <= kmax*kmax)) Nf++;

	Kf = (int *)malloc((3 * Nf) * sizeof(int));
	abc = (hipfftDoubleReal *)malloc((3) * sizeof(hipfftDoubleReal));

	Nf = 0;
	for (int ix = 1; ix < imax; ix += 1)
		for (int iy = 1; iy < imax; iy += 1)
			for (int iz = 1; iz < imax; iz += 1)
				if ((ix*ix + iy*iy + iz*iz >= kmin*kmin) && (ix*ix + iy*iy + iz*iz <= kmax*kmax))
				{
					Kf[3 * Nf] = ix;
					Kf[3 * Nf + 1] = iy;
					Kf[3 * Nf + 2] = iz;
					Nf++;
				}

	printf("made data for making specters great again\n");


				///////////////////////////////////////////////
				//если N=256, мы храним в памяти видяхи, если больше, используем оперативку
				///////////////////////////////////////////////
	if (N>256) {
	hipHostAlloc((void **)&h_UX, csize,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&UX,h_UX,0);

	hipHostAlloc((void **)&h_UY, csize,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&UY,h_UY,0);

	hipHostAlloc((void **)&h_UZ, csize,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&UZ,h_UZ,0);

	hipHostAlloc((void **)&h_UXv, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&UXv,h_UXv,0);

	hipHostAlloc((void **)&h_UYv, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&UYv,h_UYv,0);

	hipHostAlloc((void **)&h_UZv, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&UZv,h_UZv,0);

	hipHostAlloc((void **)&h_BX, csize,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&BX,h_BX,0);

	hipHostAlloc((void **)&h_BY, csize,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&BY,h_BY,0);

	hipHostAlloc((void **)&h_BZ, csize,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&BZ,h_BZ,0);

	hipHostAlloc((void **)&h_BXv, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&BXv,h_BXv,0);

	hipHostAlloc((void **)&h_BYv, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&BYv,h_BYv,0);

	hipHostAlloc((void **)&h_BZv, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&BZv,h_BZv,0);


	//кусок кода под всякие там разные сервисные переменные
	hipMalloc((void **)&dUv, csizeIm);

	hipHostAlloc((void **)&h_AXv, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&AXv,h_AXv,0);

	hipHostAlloc((void **)&h_AYv, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&AYv,h_AYv,0);

	hipHostAlloc((void **)&h_AZv, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&AZv,h_AZv,0);

	hipMalloc((void **)&d1U, csize);

	//вернуть, если памяти хватит
    //hipMalloc((void **)&d2U, csize);
	hipHostAlloc((void **)&h_d2U, csize,
		hipHostMallocWriteCombined |
		hipHostMallocMapped);
	hipHostGetDevicePointer(&d2U, h_d2U, 0);

	hipHostAlloc((void **)&h_d3U, csize,
		hipHostMallocWriteCombined |
		hipHostMallocMapped);
	hipHostGetDevicePointer(&d3U, h_d3U, 0);

	hipHostAlloc((void **)&h_d4U, csize,
		hipHostMallocWriteCombined |
		hipHostMallocMapped);
	hipHostGetDevicePointer(&d4U, h_d4U, 0);

	hipHostAlloc((void **)&h_d5U, csize,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&d5U,h_d5U,0);

	hipHostAlloc((void **)&h_d6U, csize,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&d6U,h_d6U,0);

	hipMalloc((void **)&AiD, csize);

	hipHostAlloc((void **)&h_P, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&P,h_P,0);

	hipHostAlloc((void **)&h_F, csizeIm,
		hipHostMallocWriteCombined |
		hipHostMallocMapped);
	hipHostGetDevicePointer(&F, h_F, 0);

	hipHostAlloc((void **)&h_U1, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&U1,h_U1,0);

	hipHostAlloc((void **)&h_U2, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&U2,h_U2,0);

	hipHostAlloc((void **)&h_U3, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&U3,h_U3,0);

	hipHostAlloc((void **)&h_U1n, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);

	hipHostGetDevicePointer(&U1n,h_U1n,0);
	hipHostAlloc((void **)&h_U2n, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&U2n,h_U2n,0);

	hipHostAlloc((void **)&h_U3n, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&U3n,h_U3n,0);

	hipHostAlloc((void **)&h_B1, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&B1,h_B1,0);

	hipHostAlloc((void **)&h_B2, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&B2,h_B2,0);

	hipHostAlloc((void **)&h_B3, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&B3,h_B3,0);

	hipHostAlloc((void **)&h_B1n, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&B1n,h_B1n,0);

	hipHostAlloc((void **)&h_B2n, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&B2n,h_B2n,0);

	hipHostAlloc((void **)&h_B3n, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&B3n,h_B3n,0);

	hipMalloc((void **)&TCH, 3 * sizeof(hipfftDoubleReal));
}
else
{

	h_UX=(hipfftDoubleReal *)malloc(csize);
	hipMalloc((void **)&UX,csize);
	h_UY=(hipfftDoubleReal *)malloc(csize);
	hipMalloc((void **)&UY,csize);
	h_UZ=(hipfftDoubleReal *)malloc(csize);
	hipMalloc((void **)&UZ,csize);

	hipMalloc((void **)&UXv,csizeIm);
	hipMalloc((void **)&UYv,csizeIm);
	hipMalloc((void **)&UZv,csizeIm);

	h_BX=(hipfftDoubleReal *)malloc(csize);
	hipMalloc((void **)&BX,csize);
	h_BY=(hipfftDoubleReal *)malloc(csize);
	hipMalloc((void **)&BY,csize);
	h_BZ=(hipfftDoubleReal *)malloc(csize);
	hipMalloc((void **)&BZ,csize);

	hipMalloc((void **)&BXv,csizeIm);
	hipMalloc((void **)&BYv,csizeIm);
	hipMalloc((void **)&BZv,csizeIm);

	//кусок кода под всякие там разные сервисные переменные
	hipMalloc((void **)&dUv, csizeIm);

	hipMalloc((void **)&AXv, csizeIm);
	hipMalloc((void **)&AYv, csizeIm);
	hipMalloc((void **)&AZv, csizeIm);

	hipMalloc((void **)&d1U, csize);
    hipMalloc((void **)&d2U, csize);
    hipMalloc((void **)&d3U, csize);
    hipMalloc((void **)&d4U, csize);
    hipMalloc((void **)&d5U, csize);
    hipMalloc((void **)&d6U, csize);

	hipMalloc((void **)&AiD, csize);

	hipMalloc((void **)&P, csizeIm);
	hipMalloc((void **)&F, csizeIm);


	hipMalloc((void **)&U1, csizeIm);
	hipMalloc((void **)&U2, csizeIm);
	hipMalloc((void **)&U3, csizeIm);

	hipMalloc((void **)&U1n, csizeIm);
	hipMalloc((void **)&U2n, csizeIm);
	hipMalloc((void **)&U3n, csizeIm);


	hipMalloc((void **)&B1, csizeIm);
	hipMalloc((void **)&B2, csizeIm);
	hipMalloc((void **)&B3, csizeIm);

	hipMalloc((void **)&B1n, csizeIm);
	hipMalloc((void **)&B2n, csizeIm);
	hipMalloc((void **)&B3n, csizeIm);

	hipMalloc((void **)&TCH, 3 * sizeof(hipfftDoubleReal));
}
	tch = (hipfftDoubleReal *)malloc(3 * sizeof(hipfftDoubleReal));
	dt = 0.;//чтобы вышел сразу же, если не посчитает dt
	tch[0] = dt;
	tch[1] = Cfl;
	tch[2] = h;
	hipMemcpy(TCH, tch, 3 * sizeof(hipfftDoubleReal), hipMemcpyHostToDevice);
	if (N>256) {
	hipHostAlloc((void **)&h_DXv, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&DXv,h_DXv,0);

	hipHostAlloc((void **)&h_DYv, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&DYv,h_DYv,0);

	hipHostAlloc((void **)&h_DZv, csizeIm,
				   hipHostMallocWriteCombined |
				   hipHostMallocMapped);
	hipHostGetDevicePointer(&DZv,h_DZv,0);
}
else
{

	hipMalloc((void **)&DXv, csizeIm);
	hipMalloc((void **)&DYv, csizeIm);
	hipMalloc((void **)&DZv, csizeIm);

}
	NullREAL <<<dimGrid, dimBlock >>> (BX);
	NullREAL <<<dimGrid, dimBlock >>> (BY);
	NullREAL <<<dimGrid, dimBlock >>> (BZ);
	NullREAL <<<dimGrid, dimBlock >>> (UX);
	NullREAL <<<dimGrid, dimBlock >>> (UY);
	NullREAL <<<dimGrid, dimBlock >>> (UZ);

	//задаём в фурье
	hipfftExecD2Z(plan, BX, BXv);
	hipfftExecD2Z(plan, BY, BYv);
	hipfftExecD2Z(plan, BZ, BZv);

	hipDeviceSynchronize();


				///////////////////////////////////////////////
				//задание начальных условий
				///////////////////////////////////////////////

	if (numberofpreviousforcing>0) for(int i=0;i<numberofpreviousforcing;i++)
	{
		phix = 2 * M_PI*double(rand()) / double(RAND_MAX);
		phiy = 2 * M_PI*double(rand()) / double(RAND_MAX);
		phiz = 2 * M_PI*double(rand()) / double(RAND_MAX);
		num1 = rand() % Nf;//число определяющее группу векторов(4 симметричных вектора), на которых форсируем
		num2 = rand() % 4;//определяем, kx,ky или -kx,ky или kx,-ky  или -kx,-ky
		if (num2 == 0) {
			kx = Kf[3 * num1];
			ky = Kf[3 * num1 + 1];
			kz = Kf[3 * num1 + 2];
		}
		else if (num2 == 1) {
			kx = -Kf[3 * num1];
			ky = Kf[3 * num1 + 1];
			kz = Kf[3 * num1 + 2];
		}
		else if (num2 == 2) {
			kx = Kf[3 * num1];
			ky = -Kf[3 * num1 + 1];
			kz = Kf[3 * num1 + 2];
		}
		else if (num2 == 3) {
			kx = -Kf[3 * num1];
			ky = -Kf[3 * num1 + 1];
			kz = Kf[3 * num1 + 2];
		}
			//гаусс, сигма=0.5, медиана - среднее арифм он Кмакс и Кмин
		//	double sigma=0.7;
		// kmod=sqrt(double(kx)*double(kx) + double(ky)*double(ky) + double(kz)*double(kz));
		// Estep = 100.0 * EForse / kmod * exp(-(kmod-0.5*double(kmax+kmin))*(kmod-0.5*double(kmax+kmin))/(2.0*sigma*sigma))/(sigma*sqrt(2.0*double(M_PI)));



		//плоский спектр
		Estep = 100.0 * EForse / sqrt(double(kx)*double(kx) + double(ky)*double(ky) + double(kz)*double(kz));

		//~k^(-2)
		//kmod=sqrt(double(kx)*double(kx) + double(ky)*double(ky) + double(kz)*double(kz));
		//Estep = 100.0 * (EForse / kmod) / (kmod*kmod);

		//~k^(-3/2?)
		//kmod=sqrt(double(kx)*double(kx) + double(ky)*double(ky) + double(kz)*double(kz));
		//Estep = 100.0 * (EForse / kmod) / (kmod*sqrt(kmod));



		FindABC(kx, ky, kz, phix, phiy, phiz, Estep, abc);//(kx,ky,kz,phi1,phi2,phi3,Eforcing,ABC)

		if (i%50==0) printf("A = %f B = %f C = %f n = %d \n",abc[0],abc[1],abc[2],i);

		/*cout << numofforsing[0] << " " << numofforsing[1] << " " << numofforsing[2] << endl;																															cout << forseandphi[0] << " " << forseandphi[1] << " " << forseandphi[2] << " " << forseandphi[3] << endl;*/
		ABC1 = abc[0];
		ABC2 = abc[1];
		ABC3 = abc[2];

		Forcing << <dimGrid, dimBlock >> > (AiD, UX, UY, UZ, kx, ky, kz, phix, phiy, phiz, ABC1, ABC2, ABC3);

		hipDeviceSynchronize();

	}

				///////////////////////////////////////////////
				//вывод начальных условий
				///////////////////////////////////////////////

	hipfftExecD2Z(plan, UX, UXv);
	hipfftExecD2Z(plan, UY, UYv);
	hipfftExecD2Z(plan, UZ, UZv);

	hipDeviceSynchronize();

				///////////////////////////////////////////////
				//вывод в физ пространстве
				///////////////////////////////////////////////
if (vivodphis)
	{
		start_time = clock()/CLOCKS_PER_SEC;

		
		strcpy(file, OutputDirectory);
		strncat(file, "UXout0.dat", 80);
		out1 = fopen(file, "w+");

		strcpy(file, OutputDirectory);
		strncat(file, "UYout0.dat",80);
		out2 = fopen(file, "w+");

		strcpy(file, OutputDirectory);
		strncat(file, "UZout0.dat",80);
		out3 = fopen(file, "w+");

        if (N<=256) {
			hipMemcpy(h_UX, UX, csize, hipMemcpyDeviceToHost);
			hipMemcpy(h_UY, UY, csize, hipMemcpyDeviceToHost);
			hipMemcpy(h_UZ, UZ, csize, hipMemcpyDeviceToHost);
			}
		for (int i = 0; i < N; i++)
			for (int j = 0; j < N; j++) {
				for (int k = 0; k < N; k++)
				{
					fprintf(out1, "%.16e ", h_UX[(i*N + j)*N + k]);
					fprintf(out2, "%.16e ", h_UY[(i*N + j)*N + k]);
					fprintf(out3, "%.16e ", h_UZ[(i*N + j)*N + k]);
				}
				fprintf(out1, "\n");
				fprintf(out2, "\n");
				fprintf(out3, "\n");
			}

		fclose(out1);
		fclose(out2);
		fclose(out3);

		strcpy(file, OutputDirectory);
		strncat(file, "BXout0.dat",80);
		out1 = fopen(file, "w+");

		strcpy(file, OutputDirectory);
		strncat(file, "BYout0.dat",80);
		out2 = fopen(file, "w+");

		strcpy(file, OutputDirectory);
		strncat(file, "BZout0.dat",80);
		out3 = fopen(file, "w+");

         if (N<=256) {
			hipMemcpy(h_BX, BX, csize, hipMemcpyDeviceToHost);
			hipMemcpy(h_BY, BY, csize, hipMemcpyDeviceToHost);
			hipMemcpy(h_BZ, BZ, csize, hipMemcpyDeviceToHost);
			}
		for (int i = 0; i < N; i++)
			for (int j = 0; j < N; j++)
			{
				for (int k = 0; k < N; k++)
				{
					fprintf(out1, "%.16e ", h_BX[(i*N + j)*N + k]);
					fprintf(out2, "%.16e ", h_BY[(i*N + j)*N + k]);
					fprintf(out3, "%.16e ", h_BZ[(i*N + j)*N + k]);
				}
				fprintf(out1, "\n");
				fprintf(out2, "\n");
				fprintf(out3, "\n");
			}

		fclose(out1);
		fclose(out2);
		fclose(out3);

		end_time = clock()/CLOCKS_PER_SEC;//конечное время ввода данных

		cout << "Raw data transfer to disk made in " << double(end_time - start_time) << " sek" << endl;
	}


				///////////////////////////////////////////////
				//вывод спектров
				///////////////////////////////////////////////

	if (vivodspec)
	{
		start_time = clock()/CLOCKS_PER_SEC;
		Sumforen << <dimGridZ, dimBlock >> > (UXv, Ukx);
		Sumforen << <dimGridZ, dimBlock >> > (UYv, Uky);
		Sumforen << <dimGridZ, dimBlock >> > (UZv, Ukz);
		hipDeviceSynchronize();

		hipMemcpy(ukx, h_Ukx, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);
		hipMemcpy(uky, h_Uky, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);
		hipMemcpy(ukz, h_Ukz, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);

		hipDeviceSynchronize();
		Sumforen << <dimGridZ, dimBlock >> > (BXv, Ukx);
		Sumforen << <dimGridZ, dimBlock >> > (BYv, Uky);
		Sumforen << <dimGridZ, dimBlock >> > (BZv, Ukz);
		hipDeviceSynchronize();

		hipMemcpy(bkx, h_Ukx, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);
		hipMemcpy(bky, h_Uky, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);
		hipMemcpy(bkz, h_Ukz, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);

		hipDeviceSynchronize();
		Null1(ex, Nl);
		Null1(ey, Nl);
		Null1(ez, Nl);
		Null1(ebx, Nl);
		Null1(eby, Nl);
		Null1(ebz, Nl);

		for (int i = 0; i < Nl; i++)
			for (int ix = 0; ix < N / 3; ix += 1)
				for (int iy = 0; iy < N / 3; iy += 1)
					for (int iz = 0; iz < N / 3; iz += 1)
						if (WN[i] == ix*ix + iy*iy + iz*iz)
						{
							id1 = (ix*N + iy)*(N / 2 + 1) + iz;
							id2 = ((-ix + N)*N + iy)*(N / 2 + 1) + iz;
							id3 = (ix*N + (-iy + N))*(N / 2 + 1) + iz;
							id4 = ((-ix + N)*N + (-iy + N))*(N / 2 + 1) + iz;
							ex[i] += ukx[id1] + ukx[id2] + ukx[id3] + ukx[id4];
							ey[i] += uky[id1] + uky[id2] + uky[id3] + uky[id4];
							ez[i] += ukz[id1] + ukz[id2] + ukz[id3] + ukz[id4];
							ebx[i] += bkx[id1] + bkx[id2] + bkx[id3] + bkx[id4];
							eby[i] += bky[id1] + bky[id2] + bky[id3] + bky[id4];
							ebz[i] += bkz[id1] + bkz[id2] + bkz[id3] + bkz[id4];
						}

		ex[0] /= 4.0;
		ey[0] /= 4.0;
		ez[0] /= 4.0;
		ebx[0] /= 4.0;
		eby[0] /= 4.0;
		ebz[0] /= 4.0;
		
		strcpy(file, OutputDirectory);
		strncat(file, "UXsp0.dat",80);
		out1 = fopen(file, "w+");

		strcpy(file, OutputDirectory);
		strncat(file, "UYsp0.dat",80);
		out2 = fopen(file, "w+");

		strcpy(file, OutputDirectory);
		strncat(file, "UZsp0.dat",80);
		out3 = fopen(file, "w+");

		for (int i = 0; i < Nl; i++)
		{
			fprintf(out1, "%.8e\n", ex[i]);
		}
		for (int i = 0; i < Nl; i++)
		{
			fprintf(out2, "%.8e\n", ey[i]);
		}
		for (int i = 0; i < Nl; i++)
		{
			fprintf(out3, "%.8e\n", ez[i]);
		}
		fclose(out1);
		fclose(out2);
		fclose(out3);

		strcpy(file, OutputDirectory);
		strncat(file, "BXsp0.dat",80);
		out1 = fopen(file, "w+");

		strcpy(file, OutputDirectory);
		strncat(file, "BYsp0.dat",80);
		out2 = fopen(file, "w+");

		strcpy(file, OutputDirectory);
		strncat(file, "BZsp0.dat",80);
		out3 = fopen(file, "w+");

		for (int i = 0; i < Nl; i++)
		{
			fprintf(out1, "%.8e\n", ebx[i]);
		}
		for (int i = 0; i < Nl; i++)
		{
			fprintf(out2, "%.8e\n", eby[i]);
		}
		for (int i = 0; i < Nl; i++)
		{
			fprintf(out3, "%.8e\n", ebz[i]);
		}
		fclose(out1);
		fclose(out2);
		fclose(out3);


		end_time = clock()/CLOCKS_PER_SEC;//конечное время ввода данных
		cout << "Calculation of spectres and data transfer to disk made in " << double(end_time - start_time) << " sek" << endl;
	}
				///////////////////////////////////////////////
				//вывод спиральностей
				///////////////////////////////////////////////


	if (vivodhelicity)
	{
		start_time = clock()/CLOCKS_PER_SEC;
				///////////////////////////////////////////////
				// расчет перекрестной спиральности
				///////////////////////////////////////////////
		CutN3 << <dimGridZ, dimBlock >> > (UXv);
		CutN3 << <dimGridZ, dimBlock >> > (UYv);
		CutN3 << <dimGridZ, dimBlock >> > (UZv);
				hipDeviceSynchronize();

		hipfftExecZ2D(planinverse, UXv, UX);
		hipfftExecZ2D(planinverse, UYv, UY);
		hipfftExecZ2D(planinverse, UZv, UZ);
				hipDeviceSynchronize();

		CutN3 << <dimGridZ, dimBlock >> > (BXv);
		CutN3 << <dimGridZ, dimBlock >> > (BYv);
		CutN3 << <dimGridZ, dimBlock >> > (BZv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, BXv, BX);
		hipfftExecZ2D(planinverse, BYv, BY);
		hipfftExecZ2D(planinverse, BZv, BZ);
				hipDeviceSynchronize();


	CrossHelicity <<< dimGrid, dimBlock >>> (UX, UY, UZ, BX, BY, BZ, AiD);
				hipDeviceSynchronize();
	hipfftExecD2Z(plan, AiD, DXv);
				hipDeviceSynchronize();

				///////////////////////////////////////////////
				//расчет спиральност
				///////////////////////////////////////////////

				///////////////////////////////////////////////
				//иксовая компонента(производные, умножаемые на соответствующую компоненту Ux)
				///////////////////////////////////////////////
		Ddy << < dimGridZ, dimBlock >> > (UZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (UYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();

				///////////////////////////////////////////////
				//игрековая компонента(производные, умножаемые на соответствующую компоненту Ux)
				///////////////////////////////////////////////
		Ddz << < dimGridZ, dimBlock >> > (UXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (UZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();

				///////////////////////////////////////////////
				//зетовая компонента(производные, умножаемые на соответствующую компоненту Ux)
				///////////////////////////////////////////////
		Ddx << < dimGridZ, dimBlock >> > (UYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d5U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (UXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d6U);
				hipDeviceSynchronize();

		Helicity	<<< dimGrid, dimBlock >>> (UX, UY, UZ, d1U, d2U, d3U, d4U, d5U, d6U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, AXv);
				hipDeviceSynchronize();


				///////////////////////////////////////////////
				//вывод и расчет спектров
				///////////////////////////////////////////////


		Sumforen << <dimGridZ, dimBlock >> > (AXv, Ukx);
		Sumforen << <dimGridZ, dimBlock >> > (DXv, Uky);
		hipDeviceSynchronize();

		hipMemcpy(ukx, h_Ukx, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);
		hipMemcpy(uky, h_Uky, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);

		hipDeviceSynchronize();

		Null1(ex, Nl);
		Null1(ey, Nl);


		for (int i = 0; i < Nl; i++)
			for (int ix = 0; ix < N / 3; ix += 1)
				for (int iy = 0; iy < N / 3; iy += 1)
					for (int iz = 0; iz < N / 3; iz += 1)
						if (WN[i] == ix*ix + iy*iy + iz*iz)
						{
							id1 = (ix*N + iy)*(N / 2 + 1) + iz;
							id2 = ((-ix + N)*N + iy)*(N / 2 + 1) + iz;
							id3 = (ix*N + (-iy + N))*(N / 2 + 1) + iz;
							id4 = ((-ix + N)*N + (-iy + N))*(N / 2 + 1) + iz;
							ex[i] += ukx[id1] + ukx[id2] + ukx[id3] + ukx[id4];
							ey[i] += uky[id1] + uky[id2] + uky[id3] + uky[id4];
						}

		ex[0] /= 4.0;
		ey[0] /= 4.0;

		strcpy(file, OutputDirectory);
		strncat(file, "HelicitySP0.dat",80);
		out1 = fopen(file, "w+");

		strcpy(file, OutputDirectory);
		strncat(file, "CrossHelicitySP0.dat",80);
		out2 = fopen(file, "w+");


		for (int i = 0; i < Nl; i++)
		{
			fprintf(out1, "%.8e\n", ex[i]);
		}
		for (int i = 0; i < Nl; i++)
		{
			fprintf(out2, "%.8e\n", ey[i]);
		}

		fclose(out1);
		fclose(out2);


		end_time = clock()/CLOCKS_PER_SEC;//конечное время ввода данных
		cout << "Calculation of helicity spectra and data transfer to disk made in " << double(end_time - start_time) << " sek" << endl;
	}

	int start_time2 = clock()/CLOCKS_PER_SEC;
	int end_time2;
	while (T <= Time)
	{
		//Считаем нелинейную матрицу A
		//компоненты UX,UY,UZ

		hipfftExecZ2D(planinverse, UXv, UX);
		hipfftExecZ2D(planinverse, UYv, UY);
		hipfftExecZ2D(planinverse, UZv, UZ);
				hipDeviceSynchronize();

		// считаем производные UX, UY, UZ, необходимые для AX и перекидываем в физ пр-во
		Ddx << < dimGridZ, dimBlock >> > (UYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (UXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (UXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (UZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();


		//собственно AX
		nelinAx << < dimGrid, dimBlock >> > (UY, UZ, d1U, d2U, d3U, d4U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, AXv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (AXv);

		//аналогично для AY
		Ddy << < dimGridZ, dimBlock >> > (UZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (UYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (UYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (UXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();

		nelinAy << < dimGrid, dimBlock >> > (UZ, UX, d1U, d2U, d3U, d4U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, AYv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (AYv);
				hipDeviceSynchronize();
		//аналогично для AZ
		Ddz << < dimGridZ, dimBlock >> > (UXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (UZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (UZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (UYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();

		nelinAz << < dimGrid, dimBlock >> > (UX, UY, d1U, d2U, d3U, d4U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, AZv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (AZv);
				hipDeviceSynchronize();



		//Аналогичные расчёты нелинейного члена D(чисто магнитного)
		//DX
		CutN3 << <dimGridZ, dimBlock >> > (BXv);
		CutN3 << <dimGridZ, dimBlock >> > (BYv);
		CutN3 << <dimGridZ, dimBlock >> > (BZv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, BXv, BX);
		hipfftExecZ2D(planinverse, BYv, BY);
		hipfftExecZ2D(planinverse, BZv, BZ);
				hipDeviceSynchronize();

		Ddx << < dimGridZ, dimBlock >> > (BYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (BXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (BXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (BZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();


		nelinDx << < dimGrid, dimBlock >> > (BY, BZ, d1U, d2U, d3U, d4U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, DXv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (DXv);
				hipDeviceSynchronize();

		////////////////////////////////////////////////////////////////////////////////////////////////////////
		//DY
		Ddy << < dimGridZ, dimBlock >> > (BZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (BYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (BYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (BXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();

		nelinDy << < dimGrid, dimBlock >> > (BZ, BX, d1U, d2U, d3U, d4U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, DYv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (DYv);
				hipDeviceSynchronize();

		////////////////////////////////////////////////////////////////////////////////////////////////////////
		// DZ
		Ddz << < dimGridZ, dimBlock >> > (BXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (BZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (BZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (BYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();

		nelinDz << < dimGrid, dimBlock >> > (BX, BY, d1U, d2U, d3U, d4U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, DZv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (DZv);
				hipDeviceSynchronize();

		//дальше надо посчитать давление ч/з лапласа

		lapP << <dimGridZ, dimBlock >> > (AXv, AYv, AZv, DXv, DYv, DZv, P);

				hipDeviceSynchronize();

		AbsZn << <dimGrid, dimBlock >> > (UX, UY, UZ); //работаем с модулями, ищем их в этой процедурe
				hipDeviceSynchronize();

		DT << <dimGrid, dimBlock >> > (UX, UY, UZ, TCH); //алгоритм поиска максимального значения на куде(редукция) и высчитывание dt
				hipDeviceSynchronize();


		hipMemcpy(tch, TCH, 3 * sizeof(hipfftDoubleReal), hipMemcpyDeviceToHost);
		if (tch[0] > dtmax) tch[0] = dtmax;
		if (tch[0] > Cfl2*h*h / nu) tch[0] = Cfl2*h*h / nu;
		dt = tch[0];
		AbsZnB << <dimGrid, dimBlock >> > (BX, BY, BZ);
				hipDeviceSynchronize();
		DT << <dimGrid, dimBlock >> > (BX, BY, BZ, TCH);
				hipDeviceSynchronize();
		hipMemcpy(tch, TCH, 3 * sizeof(hipfftDoubleReal), hipMemcpyDeviceToHost);
		if (tch[0] > dt) tch[0] = dt;
		printf("%lf %lf \n", tch[0], T + tch[0]);
		hipMemcpy(TCH, tch, 3 * sizeof(hipfftDoubleReal), hipMemcpyHostToDevice);


		//осталось только посчитать F(правая часть уравнения)

		Fx << <dimGridZ, dimBlock >> > (AXv, UXv, UYv, UZv, P, F, DXv);
				hipDeviceSynchronize();
		// тут считается мацуно для Fx

		Mazuno << <dimGridZ, dimBlock >> > (UXv, U1, F, TCH);
				hipDeviceSynchronize();

				Multiple << <dimGridZ, dimBlock >> > (U1, U1n);
				hipDeviceSynchronize();

		Mazuno << <dimGridZ, dimBlock >> > (U1n, U1, F, TCH);
				hipDeviceSynchronize();

		//теперь считаем Fy и мацуно для Fy

		Fy << <dimGridZ, dimBlock >> > (AYv, UXv, UYv, UZv, P, F, DYv);
				hipDeviceSynchronize();

		// мацуно Fy

		Mazuno << <dimGridZ, dimBlock >> > (UYv, U2, F, TCH);
				hipDeviceSynchronize();

				Multiple << <dimGridZ, dimBlock >> > (U2, U2n);

				hipDeviceSynchronize();
		Mazuno << <dimGridZ, dimBlock >> > (U2n, U2, F, TCH);
				hipDeviceSynchronize();

		// теперь считаем Fz и мацуно для Fz

		Fz << <dimGridZ, dimBlock >> > (AZv, UXv, UYv, UZv, P, F, DZv);
				hipDeviceSynchronize();

		// мацуно Fz

		Mazuno << <dimGridZ, dimBlock >> > (UZv, U3, F, TCH);
				hipDeviceSynchronize();

				Multiple << <dimGridZ, dimBlock >> > (U3, U3n);
				hipDeviceSynchronize();
		Mazuno << <dimGridZ, dimBlock >> > (U3n, U3, F, TCH);
				hipDeviceSynchronize();

		hipfftExecZ2D(planinverse, UXv, UX);
		hipfftExecZ2D(planinverse, UYv, UY);
		hipfftExecZ2D(planinverse, UZv, UZ);
		hipfftExecZ2D(planinverse, BXv, BX);
		hipfftExecZ2D(planinverse, BYv, BY);
		hipfftExecZ2D(planinverse, BZv, BZ);
				hipDeviceSynchronize();
		//Поиск Fb(сначала посчитать D со скоростями)//поменять расчётную функцию

		Ddx << < dimGridZ, dimBlock >> > (BXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (BXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (BXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (UXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (UXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d5U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (UXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d6U);
				hipDeviceSynchronize();

		//DX
		nelinDb << < dimGrid, dimBlock >> > (UX, UY, UZ, BX, BY, BZ, d1U, d2U, d3U, d4U, d5U, d6U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, DXv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (DXv);
				hipDeviceSynchronize();
		////////////////////////////////////////////////////////////////
		Ddx << < dimGridZ, dimBlock >> > (BYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (BYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (BYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (UYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (UYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d5U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (UYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d6U);
				hipDeviceSynchronize();
		//DY

		nelinDb << < dimGrid, dimBlock >> > (UX, UY, UZ, BX, BY, BZ, d1U, d2U, d3U, d4U, d5U, d6U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, DYv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (DYv);
				hipDeviceSynchronize();

		////////////////////////////////////////////////////////////////
		Ddx << < dimGridZ, dimBlock >> > (BZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (BZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (BZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (UZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (UZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d5U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (UZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d6U);
				hipDeviceSynchronize();
		////////////////////////////////////////////////////////////////////////////////////////////////////////

		// DZ
		nelinDb << < dimGrid, dimBlock >> > (UX, UY, UZ, BX, BY, BZ, d1U, d2U, d3U, d4U, d5U, d6U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, DZv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (DZv);
				hipDeviceSynchronize();

		//Fbx
		Fbx << <dimGridZ, dimBlock >> > (DXv, BXv, BYv, BZv, F);
				hipDeviceSynchronize();

		//мацуно Fbx
		Mazuno << <dimGridZ, dimBlock >> > (BXv, B1, F, TCH);
				hipDeviceSynchronize();

				Multiple << <dimGridZ, dimBlock >> > (B1, B1n);

				hipDeviceSynchronize();
		Mazuno << <dimGridZ, dimBlock >> > (B1n, B1, F, TCH);
				hipDeviceSynchronize();

		//Fby
		Fby << <dimGridZ, dimBlock >> > (DYv, BXv, BYv, BZv, F);
				hipDeviceSynchronize();

		//мацуно Fby
		Mazuno << <dimGridZ, dimBlock >> > (BYv, B2, F, TCH);
				hipDeviceSynchronize();

				Multiple << <dimGridZ, dimBlock >> > (B2, B2n);

				hipDeviceSynchronize();
		Mazuno << <dimGridZ, dimBlock >> > (B2n, B2, F, TCH);
				hipDeviceSynchronize();
		//Fbz
		Fbz << <dimGridZ, dimBlock >> > (DZv, BXv, BYv, BZv, F);
				hipDeviceSynchronize();

		//мацуно Fbz
		Mazuno << <dimGridZ, dimBlock >> > (BZv, B3, F, TCH);
				hipDeviceSynchronize();

				Multiple << <dimGridZ, dimBlock >> > (B3, B3n);

				hipDeviceSynchronize();
		Mazuno << <dimGridZ, dimBlock >> > (B3n, B3, F, TCH);
				hipDeviceSynchronize();






		//повторяем расчёт, чтобы найти F*, зависящее от промежуточного значения U*
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		//Считаем нелинейную матрицу A*
		//компоненты UX,UY,UZ в физ(нужны для А*)
		CutN3 << <dimGridZ, dimBlock >> > (U1);
		CutN3 << <dimGridZ, dimBlock >> > (U2);
		CutN3 << <dimGridZ, dimBlock >> > (U3);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, U1, UX);
		hipfftExecZ2D(planinverse, U2, UY);
		hipfftExecZ2D(planinverse, U3, UZ);
				hipDeviceSynchronize();

		// считаем производные UX*, UY*, UZ*, необходимые для AX и перекидываем в физ пр-во
		Ddx << < dimGridZ, dimBlock >> > (U2, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (U1, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (U1, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (U3, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();




		//собственно AX*
		nelinAx << < dimGrid, dimBlock >> > (UY, UZ, d1U, d2U, d3U, d4U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, AXv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (AXv);
				hipDeviceSynchronize();

		//аналогично для AY
		Ddy << < dimGridZ, dimBlock >> > (U3, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (U2, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (U2, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (U1, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();

		nelinAy << < dimGrid, dimBlock >> > (UZ, UX, d1U, d2U, d3U, d4U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, AYv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (AYv);
				hipDeviceSynchronize();

		//аналогично для AZ*
		Ddz << < dimGridZ, dimBlock >> > (U1, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (U3, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (U3, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (U2, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();

		nelinAz << < dimGrid, dimBlock >> > (UX, UY, d1U, d2U, d3U, d4U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, AZv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (AZv);
				hipDeviceSynchronize();

		//Аналогичные расчёты нелинейного члена D*(чисто магнитного)
		CutN3 << <dimGridZ, dimBlock >> > (B1);
		CutN3 << <dimGridZ, dimBlock >> > (B2);
		CutN3 << <dimGridZ, dimBlock >> > (B3);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, B1, BX);
		hipfftExecZ2D(planinverse, B2, BY);
		hipfftExecZ2D(planinverse, B3, BZ);
				hipDeviceSynchronize();

		Ddx << < dimGridZ, dimBlock >> > (B2, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (B1, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (B1, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (B3, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();

		//DX*
		nelinDx << < dimGrid, dimBlock >> > (BY, BZ, d1U, d2U, d3U, d4U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, DXv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (DXv);
				hipDeviceSynchronize();

		////////////////////////////////////////////////////////////////////////////////////////////////////////
		Ddy << < dimGridZ, dimBlock >> > (B3, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (B2, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (B2, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (B1, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();

		//DY*

		nelinDy << < dimGrid, dimBlock >> > (BZ, BX, d1U, d2U, d3U, d4U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, DYv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (DYv);
				hipDeviceSynchronize();

		////////////////////////////////////////////////////////////////////////////////////////////////////////

		Ddz << < dimGridZ, dimBlock >> > (B1, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (B3, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (B3, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (B2, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();

		// DZ
		nelinDz << < dimGrid, dimBlock >> > (BX, BY, d1U, d2U, d3U, d4U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, DZv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (DZv);
				hipDeviceSynchronize();
		//дальше надо посчитать давление* ч/з лапласа

		lapP << <dimGridZ, dimBlock >> > (AXv, AYv, AZv, DXv, DYv, DZv, P);
				hipDeviceSynchronize();

		//осталось только посчитать F*(правая часть уравнения)
		//Fx*
		Fx << <dimGridZ, dimBlock >> > (AXv, U1, U2, U3, P, F, DXv);
				hipDeviceSynchronize();
		//мацуно Fx*
		Mazuno << <dimGridZ, dimBlock >> > (U1n, UXv, F, TCH);
				hipDeviceSynchronize();

		//теперь считаем Fy*
		Fy << <dimGridZ, dimBlock >> > (AYv, U1, U2, U3, P, F, DYv);
				hipDeviceSynchronize();
		// мацуно Fy*
		Mazuno << <dimGridZ, dimBlock >> > (U2n, UYv, F, TCH);
				hipDeviceSynchronize();

		//теперь считаем Fz*
		Fz << <dimGridZ, dimBlock >> > (AZv, U1, U2, U3, P, F, DZv);
				hipDeviceSynchronize();
		// мацуно Fz*
		Mazuno << <dimGridZ, dimBlock >> > (U3n, UZv, F, TCH);
				hipDeviceSynchronize();


		//DX*(со скоростью)
		Ddx << < dimGridZ, dimBlock >> > (B1, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (B1, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (B1, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (U1, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (U1, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d5U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (U1, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d6U);
				hipDeviceSynchronize();

		nelinDb << < dimGrid, dimBlock >> > (UX, UY, UZ, BX, BY, BZ, d1U, d2U, d3U, d4U, d5U, d6U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, DXv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (DXv);
				hipDeviceSynchronize();
		///////////////////////////////////////
		//DY*(cо скоростью)
		Ddx << < dimGridZ, dimBlock >> > (B2, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (B2, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (B2, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (U2, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (U2, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d5U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (U2, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d6U);
				hipDeviceSynchronize();

		nelinDb << < dimGrid, dimBlock >> > (UX, UY, UZ, BX, BY, BZ, d1U, d2U, d3U, d4U, d5U, d6U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, DYv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (DYv);
				hipDeviceSynchronize();
		////////////////////////////////////////////
		//DZ*(cо скоростью)
		Ddx << < dimGridZ, dimBlock >> > (B3, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (B3, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (B3, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (U3, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (U3, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d5U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (U3, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d6U);
				hipDeviceSynchronize();

		nelinDb << < dimGrid, dimBlock >> > (UX, UY, UZ, BX, BY, BZ, d1U, d2U, d3U, d4U, d5U, d6U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, DZv);
				hipDeviceSynchronize();
		CutN3 << <dimGridZ, dimBlock >> > (DZv);
				hipDeviceSynchronize();

	//нахрена это тут??????????????????!!!!!!!!!!!!!!!!!!!!!!!(это было для отключения магнитных полей)
	//	Null << <dimGridZ, dimBlock >> > (B1n);
	//	Null << <dimGridZ, dimBlock >> > (B2n);
	//	Null << <dimGridZ, dimBlock >> > (B3n);
	//			hipDeviceSynchronize();
	//видимо, чтобы жизнь мёдом не казалась!!
		//Fbx*
		Fbx << <dimGridZ, dimBlock >> > (DXv, B1, B2, B3, F);
				hipDeviceSynchronize();

		//мацуно Fbx*
		Mazuno << <dimGridZ, dimBlock >> > (B1n, BXv, F, TCH);
				hipDeviceSynchronize();

		//Fby*
		Fby << <dimGridZ, dimBlock >> > (DYv, B1, B2, B3, F);
				hipDeviceSynchronize();

		//мацуно Fby*
		Mazuno << <dimGridZ, dimBlock >> > (B2n, BYv, F, TCH);
				hipDeviceSynchronize();

		//Fbz*
		Fbz << <dimGridZ, dimBlock >> > (DZv, B1, B2, B3, F);
				hipDeviceSynchronize();

		//мацуно Fbz*
		Mazuno << <dimGridZ, dimBlock >> > (B3n, BZv, F, TCH);
				hipDeviceSynchronize();





//Uxv, Uyv,Uzv вместо копирования копируем с добавлением форсинга после вывода данных
		T += tch[0];
		CutN3 << <dimGridZ, dimBlock >> > (UXv);
		CutN3 << <dimGridZ, dimBlock >> > (UYv);
		CutN3 << <dimGridZ, dimBlock >> > (UZv);
		CutN3 << <dimGridZ, dimBlock >> > (BXv);
		CutN3 << <dimGridZ, dimBlock >> > (BYv);
		CutN3 << <dimGridZ, dimBlock >> > (BZv);

				hipDeviceSynchronize();

			if ((vivodphis)&&(T>Nk2))
			{
				start_time = clock()/CLOCKS_PER_SEC;
				hipfftExecZ2D(planinverse, UXv, UX);
				hipfftExecZ2D(planinverse, UYv, UY);
				hipfftExecZ2D(planinverse, UZv, UZ);
				hipDeviceSynchronize();
				DNNN << <dimGrid, dimBlock >> > (UX);
				DNNN << <dimGrid, dimBlock >> > (UY);
				DNNN << <dimGrid, dimBlock >> > (UZ);
				hipDeviceSynchronize();

				strcpy(file, OutputDirectory);
				strncat(file, "UXout",80);
				sprintf(dir,"%d",NNk2);
				strncat(file, dir, 10);
				strncat(file, ".dat", 80);
				out1 = fopen(file, "w+");


				strcpy(file, OutputDirectory);
				strncat(file, "UYout",80);
				sprintf(dir,"%d",NNk2);
				strncat(file, dir, 10);
				strncat(file, ".dat", 80);
				out2 = fopen(file, "w+");


				strcpy(file, OutputDirectory);
				strncat(file, "UZout",80);
				sprintf(dir,"%d",NNk2);
				strncat(file, dir, 10);
				strncat(file, ".dat", 80);
				out3 = fopen(file, "w+");

				 if (N<=256) {
					hipMemcpy(h_UX, UX, csize, hipMemcpyDeviceToHost);
					hipMemcpy(h_UY, UY, csize, hipMemcpyDeviceToHost);
					hipMemcpy(h_UZ, UZ, csize, hipMemcpyDeviceToHost);
					}

				for (int i = 0; i < N; i++)
					for (int j = 0; j < N; j++) {
						for (int k = 0; k < N; k++)
						{
							fprintf(out1, "%.16e ", h_UX[(i*N + j)*N + k]);
							fprintf(out2, "%.16e ", h_UY[(i*N + j)*N + k]);
							fprintf(out3, "%.16e ", h_UZ[(i*N + j)*N + k]);
						}
						fprintf(out1, "\n");
						fprintf(out2, "\n");
						fprintf(out3, "\n");
					}

				fclose(out1);
				fclose(out2);
				fclose(out3);


				hipfftExecZ2D(planinverse, BXv, UX);
				hipfftExecZ2D(planinverse, BYv, UY);
				hipfftExecZ2D(planinverse, BZv, UZ);
				hipDeviceSynchronize();
				DNNN << <dimGrid, dimBlock >> > (UX);
				DNNN << <dimGrid, dimBlock >> > (UY);
				DNNN << <dimGrid, dimBlock >> > (UZ);
				hipDeviceSynchronize();


				strcpy(file, OutputDirectory);
				strncat(file, "BXout",80);
				sprintf(dir,"%d",NNk2);
				strncat(file, dir, 10);
				strncat(file, ".dat", 80);
				out1 = fopen(file, "w+");


				strcpy(file, OutputDirectory);
				strncat(file, "BYout",80);
				sprintf(dir,"%d",NNk2);
				strncat(file, dir, 10);
				strncat(file, ".dat", 80);
				out2 = fopen(file, "w+");


				strcpy(file, OutputDirectory);
				strncat(file, "BZout",80);
				sprintf(dir,"%d",NNk2);
				strncat(file, dir, 10);
				strncat(file, ".dat", 80);
				out3 = fopen(file, "w+");

				 if (N<=256) {
					hipMemcpy(h_BX, BX, csize, hipMemcpyDeviceToHost);
					hipMemcpy(h_BY, BY, csize, hipMemcpyDeviceToHost);
					hipMemcpy(h_BZ, BZ, csize, hipMemcpyDeviceToHost);
					}

				for (int i = 0; i < N; i++)
					for (int j = 0; j < N; j++)
					{
						for (int k = 0; k < N; k++)
						{
							fprintf(out1, "%.16e ", h_UX[(i*N + j)*N + k]);
							fprintf(out2, "%.16e ", h_UY[(i*N + j)*N + k]);
							fprintf(out3, "%.16e ", h_UZ[(i*N + j)*N + k]);
						}
						fprintf(out1, "\n");
						fprintf(out2, "\n");
						fprintf(out3, "\n");
					}
				fclose(out1);
				fclose(out2);
				fclose(out3);

				Nk2 += dNk2;
				NNk2 += 10;

				end_time = clock()/CLOCKS_PER_SEC;//конечное время ввода данных

				cout << "Raw data transfer to disk made in " << double(end_time - start_time) << " sek" << endl;
			}

			//specters
			if ((vivodspec) && (T>Nk))
			{
				start_time = clock()/CLOCKS_PER_SEC;
				Sumforen << <dimGridZ, dimBlock >> > (UXv, Ukx);
				Sumforen << <dimGridZ, dimBlock >> > (UYv, Uky);
				Sumforen << <dimGridZ, dimBlock >> > (UZv, Ukz);
				hipDeviceSynchronize();


				hipMemcpy(ukx, h_Ukx, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);
				hipMemcpy(uky, h_Uky, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);
				hipMemcpy(ukz, h_Ukz, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);

				hipDeviceSynchronize();
				Sumforen << <dimGridZ, dimBlock >> > (BXv, Ukx);
				Sumforen << <dimGridZ, dimBlock >> > (BYv, Uky);
				Sumforen << <dimGridZ, dimBlock >> > (BZv, Ukz);
				hipDeviceSynchronize();

				hipMemcpy(bkx, h_Ukx, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);
				hipMemcpy(bky, h_Uky, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);
				hipMemcpy(bkz, h_Ukz, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);

				hipDeviceSynchronize();
				Null1(ex, Nl);
				Null1(ey, Nl);
				Null1(ez, Nl);
				Null1(ebx, Nl);
				Null1(eby, Nl);
				Null1(ebz, Nl);


				for (int i = 0; i < Nl; i++)
					for (int ix = 0; ix < N / 3; ix += 1)
						for (int iy = 0; iy < N / 3; iy += 1)
							for (int iz = 1; iz < N / 3; iz += 1)
								if (WN[i] == ix*ix + iy*iy + iz*iz)
								{
									id1 = (ix*N + iy)*(N / 2 + 1) + iz;
									id2 = ((-ix + N)*N + iy)*(N / 2 + 1) + iz;
									id3 = (ix*N + (-iy + N))*(N / 2 + 1) + iz;
									id4 = ((-ix + N)*N + (-iy + N))*(N / 2 + 1) + iz;
									ex[i] += ukx[id1] + ukx[id2] + ukx[id3] + ukx[id4];
									ey[i] += uky[id1] + uky[id2] + uky[id3] + uky[id4];
									ez[i] += ukz[id1] + ukz[id2] + ukz[id3] + ukz[id4];
									ebx[i] += bkx[id1] + bkx[id2] + bkx[id3] + bkx[id4];
									eby[i] += bky[id1] + bky[id2] + bky[id3] + bky[id4];
									ebz[i] += bkz[id1] + bkz[id2] + bkz[id3] + bkz[id4];
								}
//убираем повторяющиеся нули
				ex[0] /= 4.0;
				ey[0] /= 4.0;
				ez[0] /= 4.0;
				ebx[0] /= 4.0;
				eby[0] /= 4.0;
				ebz[0] /= 4.0;

				strcpy(file, OutputDirectory);
				strncat(file, "UXsp",80);
				sprintf(dir,"%d",NNk);
				strncat(file, dir, 10);
				strncat(file, ".dat", 80);
				out1 = fopen(file, "w+");


				strcpy(file, OutputDirectory);
				strncat(file, "UYsp",80);
				sprintf(dir,"%d",NNk);
				strncat(file, dir, 10);
				strncat(file, ".dat", 80);
				out2 = fopen(file, "w+");


				strcpy(file, OutputDirectory);
				strncat(file, "UZsp",80);
				sprintf(dir,"%d",NNk);
				strncat(file, dir, 10);
				strncat(file, ".dat", 80);
				out3 = fopen(file, "w+");

				for (int i = 0; i < Nl; i++)
				{
					fprintf(out1, "%.8e\n", ex[i]);
				}
				for (int i = 0; i < Nl; i++)
				{
					fprintf(out2, "%.8e\n", ey[i]);
				}
				for (int i = 0; i < Nl; i++)
				{
					fprintf(out3, "%.8e\n", ez[i]);
				}
				fclose(out1);
				fclose(out2);
				fclose(out3);


				strcpy(file, OutputDirectory);
				strncat(file, "BXsp",80);
				sprintf(dir,"%d",NNk);
				strncat(file, dir, 10);
				strncat(file, ".dat", 80);
				out1 = fopen(file, "w+");


				strcpy(file, OutputDirectory);
				strncat(file, "BYsp",80);
				sprintf(dir,"%d",NNk);
				strncat(file, dir, 10);
				strncat(file, ".dat", 80);
				out2 = fopen(file, "w+");


				strcpy(file, OutputDirectory);
				strncat(file, "BZsp",80);
				sprintf(dir,"%d",NNk);
				strncat(file, dir, 10);
				strncat(file, ".dat", 80);
				out3 = fopen(file, "w+");

				for (int i = 0; i < Nl; i++)
				{
					fprintf(out1, "%.8e\n", ebx[i]);
				}
				for (int i = 0; i < Nl; i++)
				{
					fprintf(out2, "%.8e\n", eby[i]);
				}
				for (int i = 0; i < Nl; i++)
				{
					fprintf(out3, "%.8e\n", ebz[i]);
				}
				fclose(out1);
				fclose(out2);
				fclose(out3);
				Nk += dNk;
				NNk += 10;

				end_time = clock()/CLOCKS_PER_SEC;//конечное время ввода данных
				cout << "Calculation of spectres and data transfer to disk made in " << double(end_time - start_time) << " sek" << endl;
			}


		if ((vivodhelicity) && (T>Nk))
	{
		start_time = clock()/CLOCKS_PER_SEC;
				///////////////////////////////////////////////
				// расчет перекрестной спиральности
				///////////////////////////////////////////////
		CutN3 << <dimGridZ, dimBlock >> > (UXv);
		CutN3 << <dimGridZ, dimBlock >> > (UYv);
		CutN3 << <dimGridZ, dimBlock >> > (UZv);
				hipDeviceSynchronize();

		hipfftExecZ2D(planinverse, UXv, UX);
		hipfftExecZ2D(planinverse, UYv, UY);
		hipfftExecZ2D(planinverse, UZv, UZ);
				hipDeviceSynchronize();

		CutN3 << <dimGridZ, dimBlock >> > (BXv);
		CutN3 << <dimGridZ, dimBlock >> > (BYv);
		CutN3 << <dimGridZ, dimBlock >> > (BZv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, BXv, BX);
		hipfftExecZ2D(planinverse, BYv, BY);
		hipfftExecZ2D(planinverse, BZv, BZ);
				hipDeviceSynchronize();


	CrossHelicity <<< dimGrid, dimBlock >>> (UX, UY, UZ, BX, BY, BZ, AiD);
				hipDeviceSynchronize();
	hipfftExecD2Z(plan, AiD, DXv);
				hipDeviceSynchronize();

				///////////////////////////////////////////////
				//расчет спиральност
				///////////////////////////////////////////////

				///////////////////////////////////////////////
				//иксовая компонента(производные, умножаемые на соответствующую компоненту Ux)
				///////////////////////////////////////////////
		Ddy << < dimGridZ, dimBlock >> > (UZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d1U);
				hipDeviceSynchronize();
		Ddz << < dimGridZ, dimBlock >> > (UYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d2U);
				hipDeviceSynchronize();

				///////////////////////////////////////////////
				//игрековая компонента(производные, умножаемые на соответствующую компоненту Ux)
				///////////////////////////////////////////////
		Ddz << < dimGridZ, dimBlock >> > (UXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d3U);
				hipDeviceSynchronize();
		Ddx << < dimGridZ, dimBlock >> > (UZv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d4U);
				hipDeviceSynchronize();

				///////////////////////////////////////////////
				//зетовая компонента(производные, умножаемые на соответствующую компоненту Ux)
				///////////////////////////////////////////////
		Ddx << < dimGridZ, dimBlock >> > (UYv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d5U);
				hipDeviceSynchronize();
		Ddy << < dimGridZ, dimBlock >> > (UXv, dUv);
				hipDeviceSynchronize();
		hipfftExecZ2D(planinverse, dUv, d6U);
				hipDeviceSynchronize();

		Helicity	<<< dimGrid, dimBlock >>> (UX, UY, UZ, d1U, d2U, d3U, d4U, d5U, d6U, AiD);
				hipDeviceSynchronize();
		hipfftExecD2Z(plan, AiD, AXv);
				hipDeviceSynchronize();


				///////////////////////////////////////////////
				//вывод и расчет спектров
				///////////////////////////////////////////////


		Sumforen << <dimGridZ, dimBlock >> > (AXv, Ukx);
		Sumforen << <dimGridZ, dimBlock >> > (DXv, Uky);
		hipDeviceSynchronize();

		hipMemcpy(ukx, h_Ukx, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);
		hipMemcpy(uky, h_Uky, N*N*(N / 2 + 1) * sizeof(hipfftDoubleReal), hipMemcpyHostToHost);

		hipDeviceSynchronize();

		Null1(ex, Nl);
		Null1(ey, Nl);


		for (int i = 0; i < Nl; i++)
			for (int ix = 0; ix < N / 3; ix += 1)
				for (int iy = 0; iy < N / 3; iy += 1)
					for (int iz = 0; iz < N / 3; iz += 1)
						if (WN[i] == ix*ix + iy*iy + iz*iz)
						{
							id1 = (ix*N + iy)*(N / 2 + 1) + iz;
							id2 = ((-ix + N)*N + iy)*(N / 2 + 1) + iz;
							id3 = (ix*N + (-iy + N))*(N / 2 + 1) + iz;
							id4 = ((-ix + N)*N + (-iy + N))*(N / 2 + 1) + iz;
							ex[i] += ukx[id1] + ukx[id2] + ukx[id3] + ukx[id4];
							ey[i] += uky[id1] + uky[id2] + uky[id3] + uky[id4];
						}

		ex[0] /= 4.0;
		ey[0] /= 4.0;

		strcpy(file, OutputDirectory);
		strncat(file, "HelicitySP",80);
		sprintf(dir,"%d",NNk);
		strncat(file, dir, 10);
		strncat(file, ".dat", 80);
		out1 = fopen(file, "w+");

		strcpy(file, OutputDirectory);
		strncat(file, "CrossHelicitySP",80);
		sprintf(dir,"%d",NNk);
		strncat(file, dir, 10);
		strncat(file, ".dat", 80);
		out2 = fopen(file, "w+");


		for (int i = 0; i < Nl; i++)
		{
			fprintf(out1, "%.8e\n", ex[i]);
		}
		for (int i = 0; i < Nl; i++)
		{
			fprintf(out2, "%.8e\n", ey[i]);
		}

		fclose(out1);
		fclose(out2);


		end_time = clock()/CLOCKS_PER_SEC;//конечное время ввода данных
		cout << "Calculation of helicity spectra and data transfer to disk made in " << double(end_time - start_time) << " sek" << endl;
	}


		NNN++;

		if (useforcing)
		{
			hipfftExecZ2D(planinverse, UXv, UX);
			hipfftExecZ2D(planinverse, UYv, UY);
			hipfftExecZ2D(planinverse, UZv, UZ);
				hipDeviceSynchronize();
			DNNN << <dimGrid, dimBlock >> > (UX);
			DNNN << <dimGrid, dimBlock >> > (UY);
			DNNN << <dimGrid, dimBlock >> > (UZ);
				hipDeviceSynchronize();

			//start_time = clock()/CLOCKS_PER_SEC;
			phix = 2 * M_PI*double(rand()) / double(RAND_MAX);
			phiy = 2 * M_PI*double(rand()) / double(RAND_MAX);
			phiz = 2 * M_PI*double(rand()) / double(RAND_MAX);
			num1 = rand() % Nf;//число определяющее группу векторов(4 симметричных вектора), на которых форсируем
			num2 = rand() % 4;//определяем, kx,ky или -kx,ky или kx,-ky  или -kx,-ky
			if (num2 == 0) {
				kx = Kf[3 * num1];
				ky = Kf[3 * num1 + 1];
				kz = Kf[3 * num1 + 2];
			}
			else if (num2 == 1) {
				kx = -Kf[3 * num1];
				ky = Kf[3 * num1 + 1];
				kz = Kf[3 * num1 + 2];
			}
			else if (num2 == 2) {
				kx = Kf[3 * num1];
				ky = -Kf[3 * num1 + 1];
				kz = Kf[3 * num1 + 2];
			}
			else if (num2 == 3) {
				kx = -Kf[3 * num1];
				ky = -Kf[3 * num1 + 1];
				kz = Kf[3 * num1 + 2];
			}

		Estep = EForse / dtmax*tch[0] / sqrt(double(kx)*double(kx) + double(ky)*double(ky) + double(kz)*double(kz));
		FindABC(kx, ky, kz, phix, phiy, phiz, Estep, abc);//(kx,ky,kz,phi1,phi2,phi3,Eforcing,ABC)
		//cout << abc[0] << " " << abc[1] << " " << abc[2] << " " << NNN << endl;
		//cout << numofforsing[0] << " " << numofforsing[1] << " " << numofforsing[2] << endl;																															cout << forseandphi[0] << " " << forseandphi[1] << " " << forseandphi[2] << " " << forseandphi[3] << endl;*/
		ABC1 = abc[0];
		ABC2 = abc[1];
		ABC3 = abc[2];

		Forcing << <dimGrid, dimBlock >> > (AiD, UX, UY, UZ, kx, ky, kz, phix, phiy, phiz, ABC1, ABC2, ABC3);




		//конечное время ввода данных

		hipfftExecD2Z(plan, UX, UXv);
		hipfftExecD2Z(plan, UY, UYv);
		hipfftExecD2Z(plan, UZ, UZv);

	//	cout << "Forcing made in " << (end_time - start_time) << " msek" << endl;
		}
				hipDeviceSynchronize();

		end_time2 = clock()/CLOCKS_PER_SEC;

		printf("step N %d time is %lf s\n", NNN, (double(end_time2 - start_time2)));
		//cout << "step N" << NNN << " time is " << double(end_time2 - start_time2) << " s\n";
	}


	//cout << "Конец вычислений." << endl;
	//end_time = clock()/CLOCKS_PER_SEC;//конечное время расчёта
	/*out.open("D:\\Files\\6\\data.dat");
	out << "Расчёт занял " << (end_time - start_time)  << " секунд" <<" на "<< NNN << " шагов"<<endl;
	out.close();*/



	hipfftDestroy(plan);
	hipfftDestroy(planinverse);
	hipFree(TCH);
	hipHostFree(h_F);
	hipHostFree(h_P);
	hipHostFree(h_UX);
	hipHostFree(h_UY);
	hipHostFree(h_UZ);
	hipHostFree(h_UXv);
	hipHostFree(h_UYv);
	hipHostFree(h_UZv);
	hipHostFree(h_AXv);
	hipHostFree(h_AYv);
	hipHostFree(h_AZv);
	hipFree(dUv);
	hipFree(d1U);
	hipHostFree(h_d2U);
	hipHostFree(h_d3U);
	hipHostFree(h_d4U);
	hipHostFree(h_d5U);
	hipHostFree(h_d6U);

	hipHostFree(h_Ukx);
	hipHostFree(h_UYv);
	hipHostFree(h_UZv);

	hipFree(AiD);

	hipHostFree(h_BX);
	hipHostFree(h_BY);
	hipHostFree(h_BZ);
	hipHostFree(h_BXv);
	hipHostFree(h_BYv);
	hipHostFree(h_BZv);
	hipHostFree(h_DXv);
	hipHostFree(h_DYv);
	hipHostFree(h_DZv);

	hipHostFree(h_U1);
	hipHostFree(h_U2);
	hipHostFree(h_U3);
	hipHostFree(h_U1n);
	hipHostFree(h_U2n);
	hipHostFree(h_U3n);
	hipHostFree(h_B1);
	hipHostFree(h_B2);
	hipHostFree(h_B3);
	hipHostFree(h_B1n);
	hipHostFree(h_B2n);
	hipHostFree(h_B3n);
	free(tch);
	free(ukx);
	free(uky);
	free(ukz);
	free(bkx);
	free(bky);
	free(bkz);
	free(ex);
	free(ey);
	free(ez);
	free(ebx);
	free(eby);
	free(ebz);
	free(WN);
	free(Kf);
	free(abc);

	return 0;
}

